#include "hip/hip_runtime.h"



#include "SeetaCudaMath.h"
#include <hipblas.h> //cuda�Դ��⺯��  
#ifdef _WIN32
#include <intrin.h>
#else
#include <x86intrin.h>
#endif

#include "hip/hip_runtime.h"
#include ""

SeetaCudaMath::SeetaCudaMath()
{
}

__global__ void matrix_procuct_kernel(const float* A, const float* B, float* C,int n, int m, int
	 k)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < m * n)
	{

		int iM = idx / n;
		int iN = idx%n;
		if (iM < m)
		{
			float sum = 0;

			for (int i = 0; i < k; i++)
			{
				sum += B[iM * k + i] * A[iN*k
					+ i];
			}

			C[iM*n + iN] = sum;
		}
	}


}
//// C = B*A^T
//void SeetaCudaMath::matrix_procuct(int cudaDevice, const float* A, const float* B, float* C, const int n,
//	const int m, const int k)
//{
//	hipSetDevice(cudaDevice);
//
//	float *dev_a = 0;
//	float *dev_b = 0;
//	float *dev_c = 0;
//
//
//	hipMalloc((void**)&dev_a,  n * k * sizeof(float));
//	hipMemcpy(dev_a, A, n * k * sizeof(float), hipMemcpyHostToDevice);
//	hipMalloc((void**)&dev_b,  m * k * sizeof(float));
//	hipMemcpy(dev_b, B, m * k * sizeof(float), hipMemcpyHostToDevice);
//	hipMalloc((void**)&dev_c,  m * n * sizeof(float));
//
//	dim3 grid(1024, 1, 1), dimBlock(n*m / 1024 + 1);
//	matrix_procuct_kernel << <grid, dimBlock >> >(dev_a, dev_b, dev_c, n, m, k);
//	hipDeviceSynchronize();
//	hipMemcpy(C, dev_c, m * n * sizeof(float), hipMemcpyDeviceToHost);
//	hipFree(dev_a);
//	hipFree(dev_b);
//	hipFree(dev_c);
//
//}

float simd_dot(const float* x, const float* y, const long& len) {
	float inner_prod = 0.0f;
	__m128 X, Y; // 128-bit values
	__m128 acc = _mm_setzero_ps(); // set to (0, 0, 0, 0)
	float temp[4];

	long i;
	for (i = 0; i + 4 < len; i += 4) {
		X = _mm_loadu_ps(x + i); // load chunk of 4 floats
		Y = _mm_loadu_ps(y + i);
		acc = _mm_add_ps(acc, _mm_mul_ps(X, Y));
	}
	_mm_storeu_ps(&temp[0], acc); // store acc into an array
	inner_prod = temp[0] + temp[1] + temp[2] + temp[3];

	// add the remaining values
	for (; i < len; ++i) {
		inner_prod += x[i] * y[i];
	}
	return inner_prod;
}
void SeetaCudaMath::matrix_procuct_cpu(const float* A, const float* B, float* C, const int n,
	const int m, const int k) {

	const float* x = B;
	for (int i = 0, idx = 0; i < m; ++i) {
		const float* y = A;
		for (int j = 0; j < n; ++j, ++idx) {
			C[idx] = simd_dot(x, y, k);
			y += k;
		}
		x += k;
	}
}

void SeetaCudaMath::resizeImgCpu(unsigned char* src_data, int src_width, int src_height, unsigned char* dest_data, int dest_width, int dest_height)
{
	double lf_x_scl = static_cast<double>(src_width) / dest_width;
	double lf_y_Scl = static_cast<double>(src_height) / dest_height;
	  for (int y = 0; y < dest_height; y++) {
      for (int x = 0; x < dest_width; x++) {
        double lf_x_s = lf_x_scl * x;
        double lf_y_s = lf_y_Scl * y;

        int n_x_s = static_cast<int>(lf_x_s);
        n_x_s = (n_x_s <= (src_width - 2) ? n_x_s : (src_width - 2));
        int n_y_s = static_cast<int>(lf_y_s);
        n_y_s = (n_y_s <= (src_height - 2) ? n_y_s : (src_height - 2));

        double lf_weight_x = lf_x_s - n_x_s;
        double lf_weight_y = lf_y_s - n_y_s;

        double dest_val = (1 - lf_weight_y) * ((1 - lf_weight_x) *
          src_data[n_y_s * src_width + n_x_s] +
          lf_weight_x * src_data[n_y_s * src_width + n_x_s + 1]) +
          lf_weight_y * ((1 - lf_weight_x) * src_data[(n_y_s + 1) * src_width + n_x_s] +
          lf_weight_x * src_data[(n_y_s + 1) * src_width + n_x_s + 1]);

		dest_data[y * dest_width + x] = static_cast<unsigned char>(dest_val);
      }
    }
}

__global__ void resize_img_kernel(unsigned char* src_data, int src_width, int src_height, unsigned char* dest_data, int dest_width, int dest_height, float lf_x_scl, float lf_y_Scl)
{
	int y = blockIdx.x;
	int x = threadIdx.x;

	float lf_x_s = lf_x_scl * x;
	float lf_y_s = lf_y_Scl * y;

	int n_x_s = static_cast<int>(lf_x_s);
	n_x_s = (n_x_s <= (src_width - 2) ? n_x_s : (src_width - 2));
	int n_y_s = static_cast<int>(lf_y_s);
	n_y_s = (n_y_s <= (src_height - 2) ? n_y_s : (src_height - 2));

	float lf_weight_x = lf_x_s - n_x_s;
	float lf_weight_y = lf_y_s - n_y_s;

	float dest_val = (1 - lf_weight_y) * ((1 - lf_weight_x) *
		src_data[n_y_s * src_width + n_x_s] +
		lf_weight_x * src_data[n_y_s * src_width + n_x_s + 1]) +
		lf_weight_y * ((1 - lf_weight_x) * src_data[(n_y_s + 1) * src_width + n_x_s] +
		lf_weight_x * src_data[(n_y_s + 1) * src_width + n_x_s + 1]);

	dest_data[y * dest_width + x] = static_cast<unsigned char>(dest_val);
}

void SeetaCudaMath::resizeImgGpu(int cudaDevice, unsigned char* src_data, int src_width, int src_height, unsigned char* dest_data, int dest_width, int dest_height)
{
	hipSetDevice(cudaDevice);

	double lf_x_scl = static_cast<double>(src_width) / dest_width;
	double lf_y_scl = static_cast<double>(src_height) / dest_height;


	unsigned char *dev_src = 0;
	unsigned char *dev_dest = 0;

	hipMalloc((void**)&dev_src, src_width * src_height * sizeof(unsigned char));
	hipMemcpy(dev_src, src_data, src_width * src_height * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_dest, dest_width * dest_height * sizeof(unsigned char));
	dim3 grid(dest_height, 1, 1), dimBlock(dest_width);
	resize_img_kernel << <grid, dimBlock >> >(dev_src, src_width, src_height, dev_dest, dest_width, dest_height, lf_x_scl, lf_y_scl);
	hipDeviceSynchronize();
	hipMemcpy(dest_data, dev_dest, dest_width * dest_height * sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipFree(dev_src);
	hipFree(dev_dest);

}
//���
void SeetaCudaMath::AnkMutBkm(int cudaDevice, const float* A, const float* B, float* C, const int n,
	const int m, const int k)
{
	hipSetDevice(cudaDevice);

	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;


	hipMalloc((void**)&dev_a, n * k * sizeof(float));
	hipMemcpy(dev_a, A, n * k * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_b, m * k * sizeof(float));
	hipMemcpy(dev_b, B, m * k * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_c, m * n * sizeof(float));
	float alpha = 1.0;
	float beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dev_b, m, dev_a, k, &beta, dev_c, m);

	hipDeviceSynchronize();
	hipMemcpy(C, dev_c, m * n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

void SeetaCudaMath::AnkMutTBmk(int cudaDevice, const float* A, const float* B, float* C, const int n,
	const int m, const int k)
{
	hipSetDevice(cudaDevice);

	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;


	hipMalloc((void**)&dev_a, n * k * sizeof(float));
	hipMemcpy(dev_a, A, n * k * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_b, m * k * sizeof(float));
	hipMemcpy(dev_b, B, m * k * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_c, m * n * sizeof(float));
	float alpha = 1.0;
	float beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, dev_b, k, dev_a, k, &beta, dev_c, m);
	hipDeviceSynchronize();
	hipMemcpy(C, dev_c, m * n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
//C = B*A^T
void SeetaCudaMath::matrix_procuct_gpu(int cudaDevice, const float* A, const float* B, float* C, const int n,
	const int m, const int k)
{

	AnkMutTBmk(cudaDevice,B, A, C, m, n, k);
	/*hipSetDevice(cudaDevice);

	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;


	hipMalloc((void**)&dev_a, n * k * sizeof(float));
	hipMemcpy(dev_a, A, n * k * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_b, m * k * sizeof(float));
	hipMemcpy(dev_b, B, m * k * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_c, m * n * sizeof(float));
	float alpha = 1.0;
	float beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, dev_a, n, dev_b, m, &beta, dev_c, n);

	hipDeviceSynchronize();
	hipMemcpy(C, dev_c, m * n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);*/
}
